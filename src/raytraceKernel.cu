#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include <cutil_math.h>
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
	ray r;
	float xstep = 2 * tan( PI / 180.0f * fov.x ) / resolution.x;
	float ystep = 2 * tan( PI / 180.0f * fov.y ) / resolution.y;
		
	glm::vec3 right = -glm::cross( view, up );
	up = glm::cross( right, view );

	glm::vec3 botleft = view - (xstep*resolution.x/2)*right - (ystep*resolution.y/2)*up;

	glm::vec3 raycast = botleft + (float)x*xstep*right + (float)y*ystep*up;

	raycast = glm::normalize(raycast);

	r.origin = eye; r.direction = raycast;
	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors, 
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
							int* lights, int numberOfLights)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if((x<=resolution.x && y<=resolution.y)){
		ray cast = raycastFromCameraKernel( resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
		
		glm::vec3 interPoint, tmpInterPoint;
		glm::vec3 normal, tmpNormal;
		glm::vec3 dcol( 0 );
		float len = 999999.99999f, tmpLen;

		colors[index] = glm::vec3( 0 );
		int object = -1;

		for( int i = 0; i < numberOfGeoms; i ++ )
		{
			switch( geoms[i].type )
			{
			case( GEOMTYPE::CUBE ):
				tmpLen = boxIntersectionTest( geoms[i], cast, tmpInterPoint, tmpNormal );
				break;
			case( GEOMTYPE::SPHERE ):
				tmpLen = sphereIntersectionTest( geoms[i], cast, tmpInterPoint, tmpNormal );
				break;
			}
			if( tmpLen < len && tmpLen > 0 )
			{
				len = tmpLen;
				normal = tmpNormal;
				interPoint = tmpInterPoint;
				object = i;
			}
		}

		if( object == -1 )
			return;
		
		glm::vec3 light = glm::vec3( 0 );
		int matid = geoms[object].materialid;

		colors[index] = glm::vec3( 0.1 ) * materials[matid].color;

		if( materials[matid].emittance > 0 )
		{
			colors[index] = materials[matid].color;
		}
		else
		{
			ray shadowcast;
			for( int j = 0; j < numberOfLights; j ++ )
			{
				bool hasLight = true;
				//get the point to use
				glm::vec3 lightPos = multiplyMV( geoms[lights[j]].transform, glm::vec4(0,0,0,1) );
				glm::vec3 lnorm = lightPos - interPoint;


				float lDist = glm::length( lnorm );
				lnorm = glm::normalize( lnorm );

				shadowcast.direction = lnorm;
				shadowcast.origin = interPoint;

				int datmofo = -1;

				for( int i = 0; i < numberOfGeoms; i ++ )
				{
					if( i != lights[j] )
					{
						switch( geoms[i].type )
						{
						case( GEOMTYPE::CUBE ):
							tmpLen = boxIntersectionTest( geoms[i], shadowcast, tmpInterPoint, tmpNormal );
							break;
						case( GEOMTYPE::SPHERE ):
							tmpLen = sphereIntersectionTest( geoms[i], shadowcast, tmpInterPoint, tmpNormal );
							break;
						}
						if( tmpLen > 0 && tmpLen < lDist )
						{
							hasLight = false;
							datmofo = i;
							break;
						}
					}
				}
				float diffuseC = 0.1;
				if( hasLight )
				{
					diffuseC = 0.7;
				}
				float diffuse = max( (float) glm::dot( lnorm, normal ), 0.0f ) * diffuseC;
				colors[index] += materials[matid].color * materials[lights[j]].color * diffuse / (float) numberOfLights;
			}
		}
		//colors[index] = generateRandomNumberFromThread(resolution, time, x, y);
	}
}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  int numberOfLights = 0;
  int* lights = new int[numberOfGeoms];

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
	if( materials[newStaticGeom.materialid].emittance > 0 )
	{
		lights[numberOfLights] = i;
		numberOfLights ++;
	}
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material* cudamater = NULL;
  hipMalloc((void**)&cudamater, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamater, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  int* cudalights = NULL;
  hipMalloc((void**)&cudalights, numberOfLights*sizeof(int));
  hipMemcpy( cudalights, lights, numberOfLights*sizeof(int), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, cudamater, numberOfMaterials, cudalights, numberOfLights);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudamater );
  hipFree( cudalights );
  delete geomList;
  delete lights;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
